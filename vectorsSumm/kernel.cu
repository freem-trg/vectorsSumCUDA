#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>

#define BLOCKS_NUM 200
#define BLOCK_SIZE 256
#define DATA_TYPE int

__global__ void my_kernel( DATA_TYPE* v1, DATA_TYPE* v2, DATA_TYPE* out ){
	unsigned int n = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	if ( n >= 50000 ) return;
	out[n] = v1[n] + v2[n];
}

using namespace std;

int main(){
	DATA_TYPE v1[ BLOCKS_NUM * BLOCK_SIZE ];
	DATA_TYPE v2[ BLOCKS_NUM * BLOCK_SIZE ];

	for( int i = 0; i < 50000; i++ ){
		v1[ i ] = i;
		v2[ i ] = 50000 - i;
	}
	
	hipSetDevice( 0 );
	DATA_TYPE* vin1;
	DATA_TYPE* vin2;
	DATA_TYPE* out;
	
	unsigned int memory_size = sizeof( DATA_TYPE ) * BLOCKS_NUM * BLOCK_SIZE;
	hipMalloc( ( void** ) &vin1, memory_size );
	hipMalloc( ( void** ) &vin2, memory_size );
	hipMalloc( ( void** ) &out, memory_size );
	
	hipMemcpy( vin1, v1, memory_size, hipMemcpyHostToDevice );
	hipMemcpy( vin2, v2, memory_size, hipMemcpyHostToDevice );
	
	dim3 block( BLOCK_SIZE );
	dim3 grid( BLOCKS_NUM );
	
	my_kernel<<< grid, block >>>( vin1, vin2, out );
	hipDeviceSynchronize();
	hipMemcpy( v1, out, memory_size, hipMemcpyDeviceToHost );
		
	for (int i = 0; i < 5; i++) cout << v1[i] << endl;
	for (int i = 49995; i < 50000; i++) cout << v1[i] << endl;

	cin.get();

	hipFree( vin1 );
	hipFree( vin2 );
	hipFree( out );
	return 0;	
}

